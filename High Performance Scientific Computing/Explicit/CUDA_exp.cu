#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <time.h>


using namespace std;

const int n = 100;    // no of threads and DoF
const int sqn = n*n;



__global__ void add(int a, int b, int *c)
{
 *c = a + b;
 printf("ss\n");
}

__global__ void hello(double *u,double *v, double *a,double *F,double *flatK, double *flatM , double *u_old,double*v_old,double *a_old,int size,double Dt){
	int i = threadIdx.x;
	//printf("sup\n");
	//printf("%f\n",Dt );

	for(int j = 0;j<1000;j++){   // 1000 = no_intervals
		//printf("(%f)\n",v[i] );
		u[i] = u_old[i] + Dt*v_old[i] + Dt*Dt*0.5f*a_old[i];
		//printf("%f\n",u[i] );
		__syncthreads();
		if(i == 0){
			a[i] = (F[i] - flatK[i*size + i]*u[i] - flatK[i*size + i+1]*u[i+1])/flatM[i*size + i];
			
		} 
		else if(i == size-1){
			a[i] = (F[i] - flatK[i*size + i]*u[i] - flatK[i*n + (i-1)]*u[i-1])/flatM[i*size + i];
			
		}
		else{
			a[i] = (F[i] - flatK[i*size + i]*u[i] - flatK[i*size + (i-1)]*u[i-1] - flatK[i*size + (i + 1)]*u[i+1])/flatM[i*size + i];
			
		}
		__syncthreads();

		v[i] = v_old[i] + Dt*(a[i] + a_old[i])*0.5f;
		__syncthreads();
		v_old[i] = v[i];
		__syncthreads();
		u_old[i] = u[i];
		__syncthreads();
		a_old[i] = a[i];
		__syncthreads();
		
	if(i==48)						///////////   49, 50, 51
	printf("%f\n",u[i]);
	
	//*var1 = u[0];
}
	//printf("%f\n",v[i] );
	
}
int main(){
	
//cudaDeviceReset();
	clock_t time_i,time_f;
	time_i = clock();

	double Dt = 0.01f;
	int no_intervals = 1000;       /////////////////////////// dof
	//Initialiing Stiffness Array and Mass Array
	double Kx[n+1],m[n];

	for(int i=0;i<n+1;i++){
		Kx[i] = 1000;     ////////////////////////////////////    SPRING CONST
		if(i != n){
			m[i] = 1;	///////////////////////// MASS
		}
	}
	

	double *u,*v,*a,*F,*flatK,*flatM,*u_old,*v_old,*a_old;
	u = (double *)malloc(n*sizeof(double));
	v = (double *)malloc(n*sizeof(double));
	a = (double *)malloc(n*sizeof(double));
	F = (double *)malloc(n*sizeof(double));
	flatM = (double *)malloc(n*n*sizeof(double));
	flatK = (double *)malloc(n*n*sizeof(double));
	u_old = (double *)malloc(n*sizeof(double));
	v_old = (double *)malloc(n*sizeof(double));
	a_old = (double *)malloc(n*sizeof(double));


	for(int i=0;i<n;i++){
		u[i] = 0;
		if(i ==49){
			v[i] = 2;      /////////////    velocity
		}
		else{
			v[i] = 0;
		}
		a[i] = 0;
		F[i] = 0;
		u_old[i] = u[i];
		v_old[i] = v[i];
		a_old[i] = a[i];
	}


	//Mass and K Matrices
	double M[n][n],K[n][n];
	for(int i=0;i<n;i++){
		for(int j=0;j<n;j++){
			M[i][j]=0;
			K[i][j]=0;
			if(i==j){
				M[i][j]=m[i];
				K[i][j]=Kx[i]+Kx[i+1];
			}
			if(j==i-1){
				K[i][j]=-Kx[i];
			}
			if(j==i+1){
				K[i][j]=-Kx[i+1];
			}
		}
	}
	for(int j=0;j<n;j++){
			for(int k=0;k<n;k++){
				flatM[j*n+k]=M[j][k];
				flatK[j*n+k] = K[j][k];
			}
	}
	int size = n;

	//Making Device Copies of all matrices (u,v,a,u_old,v_old,a_old,F,flatM,flatK);
	double *dev_u,*dev_v,*dev_a,*dev_F,*dev_flatM,*dev_flatK,*dev_u_old,*dev_v_old,*dev_a_old,*dev_size;
	hipMalloc((void **)&dev_u,n*sizeof(double));
	hipMalloc((void **)&dev_v,n*sizeof(double));
	hipMalloc((void **)&dev_a,n*sizeof(double));
	hipMalloc((void **)&dev_F,n*sizeof(double));
	hipMalloc((void **)&dev_flatK,n*n*sizeof(double));
	hipMalloc((void **)&dev_flatM,n*n*sizeof(double));
	hipMalloc((void **)&dev_u_old,n*n*sizeof(double));
	hipMalloc((void **)&dev_v_old,n*n*sizeof(double));
	hipMalloc((void **)&dev_a_old,n*n*sizeof(double));
	hipMalloc((void **)&dev_size,sizeof(int));


	//Transfering all arrays to Device

	hipMemcpy(dev_u,u,n*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_v,v,n*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_a,a,n*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_F,F,n*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_flatK,flatK,n*n*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_flatM,flatM,n*n*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_u_old,u_old,n*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_v_old,v_old,n*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_a_old,a_old,n*sizeof(double),hipMemcpyHostToDevice);
	//cudaMemcpy(dev_size,size,sizeof(int),cudaMemcpyHostToDevice);

	//Running Kernal
	int N = 1; //Number of Blocks
	int t = n; // threads per block
	//Constraint -- N*t = n;
	int a1,b,c;
	int *dev_c;
	a1=3;	
	b=4;
	hipMalloc((void**)&dev_c, sizeof(int));
	//hello<<<1,10>>>();
	//add<<<1,10>>>(a1,b,dev_c);
	//for(int ii=0;ii<1000;ii++) 
	hello<<<1,n>>>(dev_u,dev_v,dev_a,dev_F,dev_flatK,dev_flatM,dev_u_old,dev_v_old,dev_a_old,size,Dt);
	//cudaMemcpy(&c, dev_c, sizeof(int), cudaMemcpyDeviceToHost);
	//printf("%d + %d is %d\n", a1, b, c);
	hipFree(dev_c);
	hipFree(dev_u);
	hipFree(dev_v);
	hipFree(dev_a);
	hipFree(dev_flatK);
	hipFree(dev_flatM);
	hipFree(dev_u_old);
	hipFree(dev_v_old);
	hipFree(dev_a_old);

	
	time_f = clock();
	double exTime = time_f - time_i;
	printf("Time Taken: %f\n",exTime/CLOCKS_PER_SEC);
	

	//func<<<1,512>>>(dev_u,dev_v,dev_a,dev_F,dev_flatK,dev_flatM,dev_u_old,dev_v_old,dev_a_old);
















return 0;
}

