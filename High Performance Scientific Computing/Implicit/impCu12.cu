#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>


using namespace std;

const int n = 3;
const int sqn = n*n;
const float beta = 0.5;





__global__ void add(int a, int b, int *c)
{
 *c = a + b;
 printf("ss\n");
}

__global__ void hello(double *u,double *v, double *a,double *F,double *flatK, double *flatM , double *u_old,double*v_old,double *a_old,int size,double Dt,double *new1,double *test){
	int i = threadIdx.x;
	double sum = 0;
	
	//printf("this thing %f\n",new1[i] );
	//printf("sup\n");
	//printf("%f\n",Dt );
	

	
	for(int j = 0;j<1000;j++){
		//printf("(%f)\n",v[i] );
		u[i] = u_old[i] + Dt*v_old[i] + Dt*Dt*0.5*a_old[i];
		//printf("%f\n",u[i] );
		__syncthreads();
		if(i == 0){
			a[i] = (F[i] - flatK[i*size + i]*u[i] - flatK[i*size + i+1]*u[i+1]);
			
		} 
		else if(i == size-1){
			a[i] = (F[i] - flatK[i*size + i]*u[i] - flatK[i*n + (i-1)]*u[i-1]);
			
		}
		else{
			a[i] = (F[i] - flatK[i*size + i]*u[i] - flatK[i*size + (i-1)]*u[i-1] - flatK[i*size + (i + 1)]*u[i+1]);
			
		}
		__syncthreads();

		for(int k=0;k<size;k++){
			sum = 0;
			for(int p = 0;p<size;p++){
				sum = sum + (new1[k*size + p] * a[i]);
				__syncthreads();
			}
			test[i] = sum;
			__syncthreads();
		}
		a[i] = test[i];
		__syncthreads();


		v[i] = v_old[i] + Dt*(a[i] + a_old[i])*0.5;
		__syncthreads();
		u[i] = u[i] +  Dt*Dt*0.5*(a[i]);
		__syncthreads();
		v_old[i] = v[i];
		__syncthreads();
		u_old[i] = u[i];
		__syncthreads();
		a_old[i] = a[i];
		__syncthreads();
		
	if(i==0)	
	printf("%f\n",u[i]);

	
	//*var1 = u[0];
}
	//printf("%f\n",v[i] );
	
	
}




void inverse(double a[], double b[])
{
	double temp=1.0;
	double inv[n+1][n+1],alpha[n+1], beta[n+2];
	double Tinv[n+1][n+1];
	for (int i = 1; i <= n; i++){
		for (int j = 1; j <= n; j++){
		inv[i][j] = a[(i-1)*n+(j-1)];
		}
	}
	alpha[0] = 1 ;
	for(int i=1;i<=n;i++){

		if(i==1) {alpha[i]=inv[i][i];}
		else {alpha[i]= inv[i][i]*alpha[i-1] - inv[i][i-1]*inv[i-1][i]*alpha[i-2];}
	}

     beta[n+1]=1.0;
	for(int i=n;i>=1;i--){
		if(i==n) beta[i]=inv[i][i];
		else beta[i]=inv[i][i]*beta[i+1] - inv[i+1][i]*inv[i][i+1]*beta[i+2];
	}
	//cout<<endl;

	Tinv[1][1]= 1/(inv[1][1] - (inv[2][1]*inv[1][2]*beta[3]/beta[2]));
	for(int i=2;i<=n;i++){
		if(i==n) Tinv[i][i] = 1/(inv[i][i] - (inv[i][i-1]*inv[i-1][i]*alpha[i-2]/alpha[i-1]));
		else Tinv[i][i] = 1/(inv[i][i] - (inv[i][i-1]*inv[i-1][i]*alpha[i-2]/alpha[i-1]) - (inv[i+1][i]*inv[i][i+1]*beta[i+2]/beta[i+1]));
	}

	for(int i=1;i<=n;i++){
		for(int j=1;j<=n;j++){
			if(i<j){

				for(int k=1;k<=j-i;k++) {
					temp=temp*inv[j-k][j-k+1];
				}
				Tinv[i][j]=temp*alpha[i-1]*Tinv[j][j]*pow(-1,j-i)/alpha[j-1];
				temp=1.0;
			}
			if(i>j){
				for(int k=1;k<=i-j;k++) {
					temp = temp*inv[j+k][j+k-1];
				}

		       	Tinv[i][j]=temp*beta[i+1]*Tinv[j][j]*pow(-1,i-j)/beta[j+1];	
				temp=1.0;
			}
			else continue;
		}
	}	

	for (int i = 1; i <= n; i++){
		for (int j = 1; j <= n; j++){
			b[(i-1)*n+(j-1)]= Tinv[i][j];
		}
	}

}


int main(){

	
	//hipDeviceReset();

	double Dt = 0.01;
	int no_intervals = 1000;
	//Initialiing Stiffness Array and Mass Array
	double Kx[n+1],m[n];

	for(int i=0;i<n+1;i++){
		Kx[i] = 1000;
		if(i != n){
			m[i] = 1;
		}
	}
	

	

	double *u,*v,*a,*F,*flatK,*flatM,*u_old,*v_old,*a_old,*new1,*test;
	u = (double *)malloc(n*sizeof(double));
	v = (double *)malloc(n*sizeof(double));
	a = (double *)malloc(n*sizeof(double));
	F = (double *)malloc(n*sizeof(double));
	flatM = (double *)malloc(n*n*sizeof(double));
	flatK = (double *)malloc(n*n*sizeof(double));
	u_old = (double *)malloc(n*sizeof(double));
	v_old = (double *)malloc(n*sizeof(double));
	a_old = (double *)malloc(n*sizeof(double));
	new1 = (double *)malloc(n*n*sizeof(double));
	test = (double *)malloc(n*sizeof(double));



	for(int i=0;i<n;i++){
		u[i] = 0;
		if(i ==0){
			v[i] = 2;
		}
		else{
			v[i] = 0;
		}
		a[i] = 0;
		F[i] = 0;
		u_old[i] = u[i];
		v_old[i] = v[i];
		a_old[i] = a[i];
	}

	//Mass and K Matrices
	double M[n][n],K[n][n];
	for(int i=0;i<n;i++){
		for(int j=0;j<n;j++){
			M[i][j]=0;
			K[i][j]=0;
			if(i==j){
				M[i][j]=m[i];
				K[i][j]=Kx[i]+Kx[i+1];
			}
			if(j==i-1){
				K[i][j]=-Kx[i];
			}
			if(j==i+1){
				K[i][j]=-Kx[i+1];
			}
		}
	}

	for(int j=0;j<n;j++){
			for(int k=0;k<n;k++){
				flatM[j*n+k]=M[j][k];
				flatK[j*n+k] = K[j][k];
			}
	}
	
	int size = n;
	double toInv[sqn];

	for(int i =0;i<sqn;i++){
		toInv[i] = beta*Dt*Dt*flatK[i] + flatM[i];
	}


	double new2[sqn];
	inverse(toInv,new2);
	for(int i=0;i<sqn;i++){
		//cout<<toInv[i]<<endl;
	}
	for(int i=0;i<sqn;i++){
		new1[i] = new2[i];
	}


	//Making Device Copies of all matrices (u,v,a,u_old,v_old,a_old,F,flatM,flatK);
	double *dev_u,*dev_v,*dev_a,*dev_F,*dev_flatM,*dev_flatK,*dev_u_old,*dev_v_old,*dev_a_old,*dev_size,*dev_new1,*dev_test;
	hipMalloc((void **)&dev_u,n*sizeof(double));
	hipMalloc((void **)&dev_v,n*sizeof(double));
	hipMalloc((void **)&dev_a,n*sizeof(double));
	hipMalloc((void **)&dev_F,n*sizeof(double));
	hipMalloc((void **)&dev_flatK,n*n*sizeof(double));
	hipMalloc((void **)&dev_flatM,n*n*sizeof(double));
	hipMalloc((void **)&dev_u_old,n*n*sizeof(double));
	hipMalloc((void **)&dev_v_old,n*n*sizeof(double));
	hipMalloc((void **)&dev_a_old,n*n*sizeof(double));
	hipMalloc((void **)&dev_size,sizeof(int));
	hipMalloc((void **)&dev_new1,n*n*sizeof(double));
	hipMalloc((void **)&dev_test,n*sizeof(double));


	//Transfering all arrays to Device

	hipMemcpy(dev_u,u,n*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_v,v,n*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_a,a,n*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_F,F,n*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_flatK,flatK,n*n*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_flatM,flatM,n*n*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_u_old,u_old,n*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_v_old,v_old,n*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_a_old,a_old,n*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_new1,new1,n*n*sizeof(double),hipMemcpyHostToDevice);
	//hipMemcpy(dev_size,size,sizeof(int),hipMemcpyHostToDevice);

	//Running Kernal
	int N = 1; //Number of Blocks
	int t = n; // threads per block
	//Constraint -- N*t = n;
	int a1,b,c;
	int *dev_c;
	a1=3;	
	b=4;
	hipMalloc((void**)&dev_c, sizeof(int));
	//hello<<<1,10>>>();
	//add<<<1,10>>>(a1,b,dev_c);
	hello<<<1,n>>>(dev_u,dev_v,dev_a,dev_F,dev_flatK,dev_flatM,dev_u_old,dev_v_old,dev_a_old,size,Dt,dev_new1,dev_test);
	//hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	//printf("%d + %d is %d\n", a1, b, c);
	hipFree(dev_c);
	hipFree(dev_u);
	hipFree(dev_v);
	hipFree(dev_a);
	hipFree(dev_flatK);
	hipFree(dev_flatM);
	hipFree(dev_u_old);
	hipFree(dev_v_old);
	hipFree(dev_a_old);

	//printf("djdm\n" );
	

	//func<<<1,512>>>(dev_u,dev_v,dev_a,dev_F,dev_flatK,dev_flatM,dev_u_old,dev_v_old,dev_a_old);
	


free(u);
free(v);
free(a);
free(F);
free(flatM);
free(flatK);
free(u_old);
free(v_old);
free(a_old);
free(new1);
free(test);









return 0;
}

